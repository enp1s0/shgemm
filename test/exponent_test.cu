#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cutf/memory.hpp>
#include <cutf/type.hpp>
#include <cutf/hiprand.hpp>
#include <mateval/comparison_cuda.hpp>
#include <shgemm/shgemm.hpp>
#include <hipblas.h>
#include <hipblas.h>

constexpr std::size_t min_log_DIM = 6;
constexpr std::size_t max_log_DIM = 12;
constexpr std::size_t log_DIM_interval = 2;
constexpr auto op_a = mtk::shgemm::op_n;
constexpr auto op_b = mtk::shgemm::op_n;

mtk::mateval::major_t convert_op_shgemm2mateval(
		const mtk::shgemm::operation_t op
		) {
	if (op == mtk::shgemm::op_n) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

hipblasOperation_t convert_op_shgemm2cublas(
		const mtk::shgemm::operation_t op
		) {
	if (op == mtk::shgemm::op_n) {
		return HIPBLAS_OP_N;
	}
	return HIPBLAS_OP_T;
}

mtk::mateval::major_t convert_op_cublas2mateval(
		const hipblasOperation_t op
		) {
	if (op == HIPBLAS_OP_N) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

std::string op_name_str(
		const mtk::shgemm::operation_t op
		) {
	if (op == mtk::shgemm::op_n) {
		return "N";
	}
	return "T";
}

std::string op_name_str(
		const hipblasOperation_t op
		) {
	if (op == HIPBLAS_OP_N) {
		return "N";
	}
	return "T";
}

void test_shgemm_core(
		mtk::shgemm::shgemmHandle_t shgemm_handle,
		mtk::shgemm::operation_t op_a,
		mtk::shgemm::operation_t op_b,
		const float* const a_fp32_ptr,
		const float* const b_fp32_ptr,
		const half * const b_fp16_ptr,
		float* const c_fp32_ptr,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const mtk::shgemm::tc_t compute_type
		) {
	const float alpha = 1.0f, beta = 0.0f;
	mtk::shgemm::shgemm(
			shgemm_handle,
			op_a, op_b,
			m, n, k,
			&alpha,
			a_fp32_ptr, (op_a == mtk::shgemm::op_n ? m : k),
			b_fp16_ptr, (op_b == mtk::shgemm::op_n ? k : n),
			&beta,
			c_fp32_ptr, m,
			compute_type
			);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	const auto [relative_max_error, residual] = mtk::mateval::cuda::max_relative_error_and_residual_AxB(
			m, n, k,
			convert_op_shgemm2mateval(op_a),
			convert_op_shgemm2mateval(op_b),
			mtk::mateval::col_major,
			a_fp32_ptr, (op_a == mtk::shgemm::op_n ? m : k),
			b_fp32_ptr, (op_b == mtk::shgemm::op_n ? k : n),
			c_fp32_ptr, m
			);
	std::printf("shgemm-%s,%lu,%lu,%lu,%s,%s,%e,%e\n",
			(compute_type == mtk::shgemm::tf32 ? "tf32" : "fp16"),
			m, n, k,
			op_name_str(op_a).c_str(),
			op_name_str(op_b).c_str(),
			residual,
			relative_max_error
			);
	std::fflush(stdout);
}

void test_cublas(
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_a,
		hipblasOperation_t op_b,
		const float* const a_fp32_ptr,
		const float* const b_fp32_ptr,
		float* const c_fp32_ptr,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const std::string mode
		) {
	if (mode == "tf32") {
		hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
	} else {
		hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
	}
	const float alpha = 1.0f, beta = 0.0f;
	hipblasSgemm(
			cublas_handle,
			op_a, op_b,
			m, n, k,
			&alpha,
			a_fp32_ptr, (op_a == HIPBLAS_OP_N ? m : k),
			b_fp32_ptr, (op_b == HIPBLAS_OP_N ? k : n),
			&beta,
			c_fp32_ptr, m
			);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	const auto [relative_max_error, residual] = mtk::mateval::cuda::max_relative_error_and_residual_AxB(
			m, n, k,
			convert_op_cublas2mateval(op_a),
			convert_op_cublas2mateval(op_b),
			mtk::mateval::col_major,
			a_fp32_ptr, (op_a == HIPBLAS_OP_N ? m : k),
			b_fp32_ptr, (op_b == HIPBLAS_OP_N ? k : n),
			c_fp32_ptr, m
			);

	std::printf("cublas-%s,%lu,%lu,%lu,%s,%s,%e,%e\n",
			mode.c_str(),
			m, n, k,
			op_name_str(op_a).c_str(),
			op_name_str(op_b).c_str(),
			residual,
			relative_max_error
			);
	std::fflush(stdout);
}

__global__ void convert_B_to_fp16_kernel(
		half* const fp16_ptr,  // [out]
		float* const fp32_ptr, // [in, out]
		const std::size_t N    // [in]
		) {
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= N) return;
	const auto fp16 = cutf::type::cast<half>(fp32_ptr[tid]);
	fp16_ptr[tid] = fp16;
	fp32_ptr[tid] = cutf::type::cast<float>(fp16);
}

void convert_B_to_fp16(
		half* const fp16_ptr,
		float* const fp32_ptr,
		const std::size_t N
		) {
	constexpr unsigned block_size = 256;
	convert_B_to_fp16_kernel<<<(N + block_size - 1) / block_size, block_size>>>(
			fp16_ptr,
			fp32_ptr,
			N
			);
	hipDeviceSynchronize();
}

__global__ void convert_A_exponent_dist_kernel(
		float* const fp32_ptr,  // [in, out]
		const int min_exponent, // [in]
		const int max_exponent, // [in]
		const std::size_t N     // [in]
		) {
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= N) return;
	fp32_ptr[tid] = powf(2.f, fp32_ptr[tid] * (max_exponent - min_exponent) + min_exponent);
}

void convert_A_exponent_dist(
		float* const fp32_ptr,
		const int min_exponent, // [in]
		const int max_exponent, // [in]
		const std::size_t N
		) {
	constexpr unsigned block_size = 256;
	convert_A_exponent_dist_kernel<<<(N + block_size - 1) / block_size, block_size>>>(
			fp32_ptr,
			min_exponent,
			max_exponent,
			N
			);
	hipDeviceSynchronize();
}

int main() {
	const auto max_N = 1lu << max_log_DIM;
	auto a_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);
	auto b_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);
	auto b_fp16_uptr = cutf::memory::get_device_unique_ptr<half >(max_N * max_N);
	auto c_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);

	const auto seed = 10lu;
	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));

	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), b_fp32_uptr.get(), max_N * max_N));
	convert_B_to_fp16(b_fp16_uptr.get(), b_fp32_uptr.get(), max_N * max_N);

	mtk::shgemm::shgemmHandle_t shgemm_handle;
	mtk::shgemm::create(shgemm_handle);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	std::vector<std::pair<int, int>> exponent_list;
	exponent_list.push_back(std::make_pair(-15, 14));
	exponent_list.push_back(std::make_pair(-36, -15));
	exponent_list.push_back(std::make_pair(-100, -36));

	std::printf("matrix,imp,m,n,k,op_a,op_b,residual,relative_max_error,throughput_in_tflops\n");
	std::fflush(stdout);
	for (const auto exponent_lim : exponent_list) {
		CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), a_fp32_uptr.get(), max_N * max_N));
		convert_A_exponent_dist(a_fp32_uptr.get(), exponent_lim.first, exponent_lim.second, max_N * max_N);
		const std::string matrix_name = std::to_string(exponent_lim.first) + ":" + std::to_string(exponent_lim.second);
		for (std::size_t log_M = min_log_DIM; log_M <= max_log_DIM; log_M += log_DIM_interval) {
			for (std::size_t log_N = min_log_DIM; log_N <= max_log_DIM; log_N += log_DIM_interval) {
				for (std::size_t log_K = min_log_DIM; log_K <= max_log_DIM; log_K += log_DIM_interval) {
					const auto m = 1lu << log_M;
					const auto n = 1lu << log_N;
					const auto k = 1lu << log_K;
					std::printf("%s,", matrix_name.c_str());
					test_shgemm_core(
							shgemm_handle,
							op_a,
							op_b,
							a_fp32_uptr.get(),
							b_fp32_uptr.get(),
							b_fp16_uptr.get(),
							c_fp32_uptr.get(),
							m, n, k,
							mtk::shgemm::tf32
							);
					std::printf("%s,", matrix_name.c_str());
					test_shgemm_core(
							shgemm_handle,
							op_a,
							op_b,
							a_fp32_uptr.get(),
							b_fp32_uptr.get(),
							b_fp16_uptr.get(),
							c_fp32_uptr.get(),
							m, n, k,
							mtk::shgemm::fp16
							);
					std::printf("%s,", matrix_name.c_str());
					test_cublas(
							cublas_handle,
							convert_op_shgemm2cublas(op_a),
							convert_op_shgemm2cublas(op_b),
							a_fp32_uptr.get(),
							b_fp32_uptr.get(),
							c_fp32_uptr.get(),
							m, n, k,
							"tf32"
							);
					std::printf("%s,", matrix_name.c_str());
					test_cublas(
							cublas_handle,
							convert_op_shgemm2cublas(op_a),
							convert_op_shgemm2cublas(op_b),
							a_fp32_uptr.get(),
							b_fp32_uptr.get(),
							c_fp32_uptr.get(),
							m, n, k,
							"fp32"
							);
				}
			}
		}
	}

	cublasFree(cublas_handle);
	mtk::shgemm::destroy(shgemm_handle);
}
