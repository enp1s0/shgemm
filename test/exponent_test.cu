#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cutf/memory.hpp>
#include <cutf/type.hpp>
#include <cutf/hiprand.hpp>
#include <mateval/comparison_cuda.hpp>
#include <shgemm/shgemm.hpp>
#include <hipblas.h>
#include <hipblas.h>

constexpr std::size_t test_count = 1lu << 6;
constexpr std::size_t min_log_DIM = 6;
constexpr std::size_t max_log_DIM = 12;
constexpr std::size_t log_DIM_interval = 2;
constexpr auto op_a = mtk::shgemm::op_n;
constexpr auto op_b = mtk::shgemm::op_n;

mtk::mateval::major_t convert_op_shgemm2mateval(
		const mtk::shgemm::operation_t op
		) {
	if (op == mtk::shgemm::op_n) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

hipblasOperation_t convert_op_shgemm2cublas(
		const mtk::shgemm::operation_t op
		) {
	if (op == mtk::shgemm::op_n) {
		return HIPBLAS_OP_N;
	}
	return HIPBLAS_OP_T;
}

mtk::mateval::major_t convert_op_cublas2mateval(
		const hipblasOperation_t op
		) {
	if (op == HIPBLAS_OP_N) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

std::string op_name_str(
		const mtk::shgemm::operation_t op
		) {
	if (op == mtk::shgemm::op_n) {
		return "N";
	}
	return "T";
}

std::string op_name_str(
		const hipblasOperation_t op
		) {
	if (op == HIPBLAS_OP_N) {
		return "N";
	}
	return "T";
}

void test_shgemm_core(
		mtk::shgemm::shgemmHandle_t shgemm_handle,
		mtk::shgemm::operation_t op_a,
		mtk::shgemm::operation_t op_b,
		const float* const a_fp32_ptr,
		const float* const b_fp32_ptr,
		const half * const b_fp16_ptr,
		float* const c_fp32_ptr,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const mtk::shgemm::tc_t compute_type
		) {
	const float alpha = 1.0f, beta = 0.0f;
	mtk::shgemm::shgemm(
			shgemm_handle,
			op_a, op_b,
			m, n, k,
			&alpha,
			a_fp32_ptr, (op_a == mtk::shgemm::op_n ? m : k),
			b_fp16_ptr, (op_b == mtk::shgemm::op_n ? k : n),
			&beta,
			c_fp32_ptr, m,
			compute_type
			);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	const auto [relative_max_error, residual] = mtk::mateval::cuda::max_relative_error_and_residual_AxB(
			m, n, k,
			convert_op_shgemm2mateval(op_a),
			convert_op_shgemm2mateval(op_b),
			mtk::mateval::col_major,
			a_fp32_ptr, (op_a == mtk::shgemm::op_n ? m : k),
			b_fp32_ptr, (op_b == mtk::shgemm::op_n ? k : n),
			c_fp32_ptr, m
			);

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto start_clock = std::chrono::system_clock::now();
	for (std::size_t test_c = 0; test_c < test_count; test_c++) {
	mtk::shgemm::shgemm(
			shgemm_handle,
			op_a, op_b,
			m, n, k,
			&alpha,
			a_fp32_ptr, (op_a == mtk::shgemm::op_n ? m : k),
			b_fp16_ptr, (op_b == mtk::shgemm::op_n ? k : n),
			&beta,
			c_fp32_ptr, m,
			compute_type
			);
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();
	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6 / test_count;

	const auto throughput = 2 * m * n * k / elapsed_time * 1e-12; // TFlop/s

	std::printf("shgemm-%s,%lu,%lu,%lu,%s,%s,%e,%e,%e\n",
			(compute_type == mtk::shgemm::tf32 ? "tf32" : "fp16"),
			m, n, k,
			op_name_str(op_a).c_str(),
			op_name_str(op_b).c_str(),
			residual,
			relative_max_error,
			throughput
			);
	std::fflush(stdout);
}

void test_cublas(
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_a,
		hipblasOperation_t op_b,
		const float* const a_fp32_ptr,
		const float* const b_fp32_ptr,
		float* const c_fp32_ptr,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const std::string mode
		) {
	if (mode == "tf32") {
		hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
	} else {
		hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);
	}
	const float alpha = 1.0f, beta = 0.0f;
	hipblasSgemm(
			cublas_handle,
			op_a, op_b,
			m, n, k,
			&alpha,
			a_fp32_ptr, (op_a == HIPBLAS_OP_N ? m : k),
			b_fp32_ptr, (op_b == HIPBLAS_OP_N ? k : n),
			&beta,
			c_fp32_ptr, m
			);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	const auto [relative_max_error, residual] = mtk::mateval::cuda::max_relative_error_and_residual_AxB(
			m, n, k,
			convert_op_cublas2mateval(op_a),
			convert_op_cublas2mateval(op_b),
			mtk::mateval::col_major,
			a_fp32_ptr, (op_a == HIPBLAS_OP_N ? m : k),
			b_fp32_ptr, (op_b == HIPBLAS_OP_N ? k : n),
			c_fp32_ptr, m
			);

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto start_clock = std::chrono::system_clock::now();
	for (std::size_t test_c = 0; test_c < test_count; test_c++) {
		hipblasSgemm(
				cublas_handle,
				op_a, op_b,
				m, n, k,
				&alpha,
				a_fp32_ptr, (op_a == HIPBLAS_OP_N ? m : k),
				b_fp32_ptr, (op_b == HIPBLAS_OP_N ? k : n),
				&beta,
				c_fp32_ptr, m
				);
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();
	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6 / test_count;

	const auto throughput = 2 * m * n * k / elapsed_time * 1e-12; // TFlop/s

	std::printf("cublas-%s,%lu,%lu,%lu,%s,%s,%e,%e,%e\n",
			mode.c_str(),
			m, n, k,
			op_name_str(op_a).c_str(),
			op_name_str(op_b).c_str(),
			residual,
			relative_max_error,
			throughput
			);
	std::fflush(stdout);
}

__global__ void convert_B_to_fp16_kernel(
		half* const fp16_ptr,  // [out]
		float* const fp32_ptr, // [in, out]
		const std::size_t N    // [in]
		) {
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= N) return;
	const auto fp16 = cutf::type::cast<half>(fp32_ptr[tid]);
	fp16_ptr[tid] = fp16;
	fp32_ptr[tid] = cutf::type::cast<float>(fp16);
}

void convert_B_to_fp16(
		half* const fp16_ptr,
		float* const fp32_ptr,
		const std::size_t N
		) {
	constexpr unsigned block_size = 256;
	convert_B_to_fp16_kernel<<<(N + block_size - 1) / block_size, block_size>>>(
			fp16_ptr,
			fp32_ptr,
			N
			);
	hipDeviceSynchronize();
}

int main() {
	const auto max_N = 1lu << max_log_DIM;
	auto a_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);
	auto b_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);
	auto b_fp16_uptr = cutf::memory::get_device_unique_ptr<half >(max_N * max_N);
	auto c_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);

	const auto seed = 10lu;
	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));

	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), a_fp32_uptr.get(), max_N * max_N));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), b_fp32_uptr.get(), max_N * max_N));
	convert_B_to_fp16(b_fp16_uptr.get(), b_fp32_uptr.get(), max_N * max_N);

	mtk::shgemm::shgemmHandle_t shgemm_handle;
	mtk::shgemm::create(shgemm_handle);

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	std::printf("imp,m,n,k,op_a,op_b,residual,relative_max_error,throughput_in_tflops\n");
	std::fflush(stdout);
	for (std::size_t log_M = min_log_DIM; log_M <= max_log_DIM; log_M += log_DIM_interval) {
		for (std::size_t log_N = min_log_DIM; log_N <= max_log_DIM; log_N += log_DIM_interval) {
			for (std::size_t log_K = min_log_DIM; log_K <= max_log_DIM; log_K += log_DIM_interval) {
				const auto m = 1lu << log_M;
				const auto n = 1lu << log_N;
				const auto k = 1lu << log_K;
				test_shgemm_core(
						shgemm_handle,
						op_a,
						op_b,
						a_fp32_uptr.get(),
						b_fp32_uptr.get(),
						b_fp16_uptr.get(),
						c_fp32_uptr.get(),
						m, n, k,
						mtk::shgemm::tf32
						);
				test_shgemm_core(
						shgemm_handle,
						op_a,
						op_b,
						a_fp32_uptr.get(),
						b_fp32_uptr.get(),
						b_fp16_uptr.get(),
						c_fp32_uptr.get(),
						m, n, k,
						mtk::shgemm::fp16
						);
				test_cublas(
						cublas_handle,
						convert_op_shgemm2cublas(op_a),
						convert_op_shgemm2cublas(op_b),
						a_fp32_uptr.get(),
						b_fp32_uptr.get(),
						c_fp32_uptr.get(),
						m, n, k,
						"tf32"
						);
				test_cublas(
						cublas_handle,
						convert_op_shgemm2cublas(op_a),
						convert_op_shgemm2cublas(op_b),
						a_fp32_uptr.get(),
						b_fp32_uptr.get(),
						c_fp32_uptr.get(),
						m, n, k,
						"fp32"
						);
			}
		}
	}

	cublasFree(cublas_handle);
	mtk::shgemm::destroy(shgemm_handle);
}
