#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cutf/memory.hpp>
#include <cutf/type.hpp>
#include <cutf/hiprand.hpp>
#include <cutf/cublas.hpp>
#include <mateval/comparison_cuda.hpp>
#include <gpu_monitor/gpu_monitor.hpp>
#include <shgemm/shgemm.hpp>

constexpr std::size_t min_log_DIM = 10;
constexpr std::size_t max_log_DIM = 14;
constexpr std::size_t log_DIM_interval = 1;
constexpr auto op_a = mtk::shgemm::op_n;
constexpr auto op_b = mtk::shgemm::op_n;

mtk::mateval::layout_t convert_op_shgemm2mateval(
		const mtk::shgemm::operation_t op
		) {
	if (op == mtk::shgemm::op_n) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

std::string op_name_str(
		const mtk::shgemm::operation_t op
		) {
	if (op == mtk::shgemm::op_n) {
		return "N";
	}
	return "T";
}

std::string op_name_str(
		const hipblasOperation_t op
		) {
	if (op == HIPBLAS_OP_N) {
		return "N";
	}
	return "T";
}

void test_shgemm_core(
		mtk::shgemm::shgemmHandle_t shgemm_handle,
		mtk::shgemm::operation_t op_a,
		mtk::shgemm::operation_t op_b,
		const float* const a_fp32_ptr,
		const float* const b_fp32_ptr,
		const half * const b_fp16_ptr,
		float* const c_fp32_ptr,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const mtk::shgemm::tc_t compute_type
		) {
	const float alpha = 1.0f, beta = 0.0f;
	const std::size_t measuring_time_in_sec = 10;
	mtk::shgemm::detail::kernel_level level;

	const std::size_t test_count_0 = 16;
	const auto start_clock = std::chrono::system_clock::now();
	for (std::size_t test_c = 0; test_c < test_count_0; test_c++) {
		mtk::shgemm::shgemm(
				shgemm_handle,
				op_a, op_b,
				m, n, k,
				&alpha,
				a_fp32_ptr, (op_a == mtk::shgemm::op_n ? m : k),
				b_fp16_ptr, (op_b == mtk::shgemm::op_n ? k : n),
				&beta,
				c_fp32_ptr, m,
				compute_type
				);
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();
	const auto elapsed_time_0 = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6 / test_count_0;
	const auto throughput_in_tflops = 2 * m * n * k / elapsed_time_0 * 1e-12;

	const std::size_t test_count = std::max<std::size_t>(1, measuring_time_in_sec / elapsed_time_0);

	const auto profiling_result = mtk::gpu_monitor::measure_power_consumption([&](){
			CUTF_CHECK_ERROR(hipDeviceSynchronize());
			for (std::size_t test_c = 0; test_c < test_count; test_c++) {
				level = mtk::shgemm::shgemm(
							shgemm_handle,
							op_a, op_b,
							m, n, k,
							&alpha,
							a_fp32_ptr, (op_a == mtk::shgemm::op_n ? m : k),
							b_fp16_ptr, (op_b == mtk::shgemm::op_n ? k : n),
							&beta,
							c_fp32_ptr, m,
							compute_type
						);
			}
			CUTF_CHECK_ERROR(hipDeviceSynchronize());
		}, 50);
	const auto elapsed_time = mtk::gpu_monitor::get_elapsed_time(profiling_result);
	const auto integrated_power_consumption = mtk::gpu_monitor::get_integrated_power_consumption(profiling_result);

	std::printf("%s,%lu,%lu,%lu,%s,%s,%e,%e,%e,%lu,%u\n",
			(compute_type == mtk::shgemm::fp16 ? "fp16" : "tf32"),
			m, n, k,
			op_name_str(op_a).c_str(),
			op_name_str(op_b).c_str(),
			throughput_in_tflops,
			integrated_power_consumption / elapsed_time,
			integrated_power_consumption / test_count,
			test_count,
			static_cast<unsigned>(level)
			);
	std::fflush(stdout);
}

void test_cublas_core(
		hipblasHandle_t cublas_handle,
		hipblasOperation_t op_a,
		hipblasOperation_t op_b,
		const float* const a_fp32_ptr,
		const float* const b_fp32_ptr,
		float* const c_fp32_ptr,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const std::string compute_type
		) {
	const float alpha = 1.0f, beta = 0.0f;
	const std::size_t measuring_time_in_sec = 10;

	if (compute_type == "TF32") {
		hipblasSetMathMode(cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH);
	} else {
		hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
	}

	const std::size_t test_count_0 = 16;
	const auto start_clock = std::chrono::system_clock::now();
	for (std::size_t test_c = 0; test_c < test_count_0; test_c++) {
		hipblasSgemm(
				cublas_handle,
				op_a, op_b,
				m, n, k,
				&alpha,
				a_fp32_ptr, (op_a == HIPBLAS_OP_N ? m : k),
				b_fp32_ptr, (op_b == HIPBLAS_OP_N ? k : n),
				&beta,
				c_fp32_ptr, m
				);
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();
	const auto elapsed_time_0 = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6 / test_count_0;
	const auto throughput_in_tflops = 2 * m * n * k / elapsed_time_0 * 1e-12;

	const std::size_t test_count = std::max<std::size_t>(1, measuring_time_in_sec / elapsed_time_0);

	const auto profiling_result = mtk::gpu_monitor::measure_power_consumption([&](){
			CUTF_CHECK_ERROR(hipDeviceSynchronize());
			for (std::size_t test_c = 0; test_c < test_count; test_c++) {
				hipblasSgemm(
							cublas_handle,
							op_a, op_b,
							m, n, k,
							&alpha,
							a_fp32_ptr, (op_a == HIPBLAS_OP_N ? m : k),
							b_fp32_ptr, (op_b == HIPBLAS_OP_N ? k : n),
							&beta,
							c_fp32_ptr, m
						);
			}
			CUTF_CHECK_ERROR(hipDeviceSynchronize());
		}, 50);
	const auto elapsed_time = mtk::gpu_monitor::get_elapsed_time(profiling_result);
	const auto integrated_power_consumption = mtk::gpu_monitor::get_integrated_power_consumption(profiling_result);

	std::printf("%s,%lu,%lu,%lu,%s,%s,%e,%e,%e,%lu,%u\n",
			compute_type.c_str(),
			m, n, k,
			op_name_str(op_a).c_str(),
			op_name_str(op_b).c_str(),
			throughput_in_tflops,
			integrated_power_consumption / elapsed_time,
			integrated_power_consumption / test_count,
			test_count,
			0
			);
	std::fflush(stdout);
	hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
}

__global__ void convert_B_to_fp16_kernel(
		half* const fp16_ptr,
		float* const fp32_ptr,
		const std::size_t N
		) {
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= N) return;
	const auto fp16 = cutf::type::cast<half>(fp32_ptr[tid]);
	fp16_ptr[tid] = fp16;
	fp32_ptr[tid] = cutf::type::cast<float>(fp16);
}

void convert_B_to_fp16(
		half* const fp16_ptr,
		float* const fp32_ptr,
		const std::size_t N
		) {
	constexpr unsigned block_size = 256;
	convert_B_to_fp16_kernel<<<(N + block_size - 1) / block_size, block_size>>>(
			fp16_ptr,
			fp32_ptr,
			N
			);
	hipDeviceSynchronize();
}

int main() {
	const auto max_N = 1lu << max_log_DIM;
	auto a_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);
	auto b_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);
	auto b_fp16_uptr = cutf::memory::get_device_unique_ptr<half >(max_N * max_N);
	auto c_fp32_uptr = cutf::memory::get_device_unique_ptr<float>(max_N * max_N);

	const auto seed = 10lu;
	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));

	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), a_fp32_uptr.get(), max_N * max_N));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), b_fp32_uptr.get(), max_N * max_N));
	convert_B_to_fp16(b_fp16_uptr.get(), b_fp32_uptr.get(), max_N * max_N);

	mtk::shgemm::shgemmHandle_t shgemm_handle;
	mtk::shgemm::create(shgemm_handle);
	auto cublas_handle_uptr = cutf::cublas::get_cublas_unique_ptr();

	std::printf("tc_t,m,n,k,op_a,op_b,,throughput_in_tflops,avg_power_consumption_in_W,integrated_power_consumption_in_Ws,test_count,kernel_level\n");
	std::fflush(stdout);
	for (std::size_t log_N = min_log_DIM; log_N <= max_log_DIM; log_N += log_DIM_interval) {
		const auto m = 1lu << log_N;
		const auto n = 1lu << log_N;
		const auto k = 1lu << log_N;
		test_shgemm_core(
				shgemm_handle,
				op_a,
				op_b,
				a_fp32_uptr.get(),
				b_fp32_uptr.get(),
				b_fp16_uptr.get(),
				c_fp32_uptr.get(),
				m, n, k,
				mtk::shgemm::tf32
				);
		test_shgemm_core(
				shgemm_handle,
				op_a,
				op_b,
				a_fp32_uptr.get(),
				b_fp32_uptr.get(),
				b_fp16_uptr.get(),
				c_fp32_uptr.get(),
				m, n, k,
				mtk::shgemm::fp16
				);
		test_cublas_core(
				*cublas_handle_uptr.get(),
				HIPBLAS_OP_N,
				HIPBLAS_OP_N,
				a_fp32_uptr.get(),
				b_fp32_uptr.get(),
				c_fp32_uptr.get(),
				m, n, k,
				"TF32"
				);
		test_cublas_core(
				*cublas_handle_uptr.get(),
				HIPBLAS_OP_N,
				HIPBLAS_OP_N,
				a_fp32_uptr.get(),
				b_fp32_uptr.get(),
				c_fp32_uptr.get(),
				m, n, k,
				"FP32"
				);
	}
	mtk::shgemm::destroy(shgemm_handle);
}
