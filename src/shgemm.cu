#include "hip/hip_runtime.h"
#include <shgemm/shgemm.hpp>
#include <cutf/cuda.hpp>
#include <wmma_extension/tcec/tcec.hpp>
#include <cassert>
#include "wmmae_shgemm.hpp"

namespace {
constexpr unsigned warp_size = 32;

template <unsigned SIZE, unsigned BLOCK_SIZE>
__device__ void mem_fill_zero(
		float* const ptr
		) {
	for (unsigned i = 0; i < SIZE; i += BLOCK_SIZE) {
		ptr[i + threadIdx.x] = 0.f;
	}
}

template<
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	unsigned BLOCK_SIZE,
	class TC_T
	>
__device__ void shgemm_core(
		float* const c_ptr,
		const float* const a_ptr,
		const half * const b_ptr
		) {
	constexpr unsigned num_submatrices = (SMEM_M / FRAG_M) * (SMEM_N / FRAG_N);
	static_assert(num_submatrices * warp_size % BLOCK_SIZE == 0, "the number of reg-level sub matrices must be a multiple of (BLOCK_SIZE / warp_size)");

	using A_Policy = typename mtk::wmma::tcec::detail::default_policy<TC_T, mtk::wmma::tcec::op_with_error_correction   , mtk::wmma::tcec::op_mma>::type;
	using B_Policy = typename mtk::wmma::tcec::detail::default_policy<TC_T, mtk::wmma::tcec::op_without_error_correction, mtk::wmma::tcec::op_mma>::type;

	for (unsigned matrix_id_offset = 0; matrix_id_offset < num_submatrices; matrix_id_offset += BLOCK_SIZE / warp_size) {
		const unsigned matrix_id = matrix_id_offset + (threadIdx.x / warp_size);
		const unsigned matrix_id_m = matrix_id % (SMEM_M / FRAG_M);
		const unsigned matrix_id_n = matrix_id / (SMEM_M / FRAG_M);

		mtk::wmma::tcec::fragment<nvcuda::wmma::accumulator, FRAG_M, FRAG_N, FRAG_K, TC_T, void, A_Policy> frag_c;
		mtk::wmma::tcec::fill_zero(frag_c);

		for (unsigned k = 0; k < SMEM_K; k += FRAG_K) {
			mtk::wmma::tcec::fragment<nvcuda::wmma::matrix_a, FRAG_M, FRAG_N, FRAG_K, TC_T, nvcuda::wmma::row_major, A_Policy> frag_a;
			mtk::wmma::tcec::load_matrix_sync(frag_a, a_ptr + matrix_id_m * FRAG_M * SMEM_K + k, SMEM_K);

			mtk::wmma::tcec::fragment<nvcuda::wmma::matrix_b, FRAG_M, FRAG_N, FRAG_K, TC_T, nvcuda::wmma::col_major, B_Policy> frag_b;
			mtk::wmma::tcec::load_matrix_sync(frag_b, b_ptr + matrix_id_n * FRAG_N * SMEM_K + k, SMEM_K);

			mtk::shgemm::mma_sync(frag_c, frag_a, frag_b, frag_c);
		}

		mtk::wmma::tcec::store_matrix_sync(c_ptr + matrix_id_m * FRAG_M + matrix_id_n * FRAG_N * SMEM_M, frag_c, SMEM_M, nvcuda::wmma::mem_col_major);
	}
}

template <class T, unsigned SMEM_M, unsigned SMEM_N, unsigned BLOCK_SIZE>
struct dmem_loader_n {
	__device__ void operator()(
			T* const smem_ptr,
			const std::size_t dmem_start_m, const std::size_t dmem_start_n,
			const std::size_t dmem_size_m, const std::size_t dmem_size_n,
			const T* const dmem_ptr, const std::size_t ldd
			) {
		if (dmem_start_m + SMEM_M < dmem_size_m && dmem_size_n + SMEM_N < dmem_size_n) {
			for (unsigned i_offset = 0; i_offset < SMEM_M * SMEM_N; i_offset += BLOCK_SIZE) {
				const auto i = i_offset + threadIdx.x;
				const auto m = (i % SMEM_M) + dmem_start_m;
				const auto n = (i / SMEM_M) + dmem_start_n;
				const auto dmem_index = m + n * ldd;

				smem_ptr[i] = dmem_ptr[dmem_index];
			}
		} else {
			for (unsigned i_offset = 0; i_offset < SMEM_M * SMEM_N; i_offset += BLOCK_SIZE) {
				const auto i = i_offset + threadIdx.x;
				const auto m = (i % SMEM_M) + dmem_start_m;
				const auto n = (i / SMEM_M) + dmem_start_n;
				const auto dmem_index = m + n * ldd;

				auto v = static_cast<T>(0);
				if (m <= dmem_size_m && n <= dmem_size_n) {
					v = dmem_ptr[dmem_index];
				}

				smem_ptr[i] = v;
			}
		}
	}
};

template <class T, unsigned SMEM_M, unsigned SMEM_N, unsigned BLOCK_SIZE>
struct dmem_storer_n {
	__device__ void operator()(
			T* const dmem_ptr, const std::size_t ldd,
			const std::size_t dmem_start_m, const std::size_t dmem_start_n,
			const std::size_t dmem_size_m, const std::size_t dmem_size_n,
			const T* const smem_ptr,
			const float alpha, const float beta
			) {
		if (beta == 0.f) {
			if (dmem_start_m + SMEM_M < dmem_size_m && dmem_size_n + SMEM_N < dmem_size_n) {
				for (unsigned i_offset = 0; i_offset < SMEM_M * SMEM_N; i_offset += BLOCK_SIZE) {
					const auto i = i_offset + threadIdx.x;
					const auto m = (i % SMEM_M) + dmem_start_m;
					const auto n = (i / SMEM_M) + dmem_start_n;
					const auto dmem_index = m + n * ldd;

					dmem_ptr[dmem_index] = smem_ptr[i] * alpha;
				}
			} else {
				for (unsigned i_offset = 0; i_offset < SMEM_M * SMEM_N; i_offset += BLOCK_SIZE) {
					const auto i = i_offset + threadIdx.x;
					const auto m = (i % SMEM_M) + dmem_start_m;
					const auto n = (i / SMEM_M) + dmem_start_n;
					const auto dmem_index = m + n * ldd;

					if (m >= dmem_size_m || n >= dmem_size_n) {
						continue;
					}

					dmem_ptr[dmem_index] = smem_ptr[i] * alpha;
				}
			}
		} else {
			if (dmem_start_m + SMEM_M < dmem_size_m && dmem_size_n + SMEM_N < dmem_size_n) {
				for (unsigned i_offset = 0; i_offset < SMEM_M * SMEM_N; i_offset += BLOCK_SIZE) {
					const auto i = i_offset + threadIdx.x;
					const auto m = (i % SMEM_M) + dmem_start_m;
					const auto n = (i / SMEM_M) + dmem_start_n;
					const auto dmem_index = m + n * ldd;

					dmem_ptr[dmem_index] = smem_ptr[i] * alpha + dmem_ptr[dmem_index] * beta;
				}
			} else {
				for (unsigned i_offset = 0; i_offset < SMEM_M * SMEM_N; i_offset += BLOCK_SIZE) {
					const auto i = i_offset + threadIdx.x;
					const auto m = (i % SMEM_M) + dmem_start_m;
					const auto n = (i / SMEM_M) + dmem_start_n;
					const auto dmem_index = m + n * ldd;

					if (m >= dmem_size_m || n >= dmem_size_n) {
						continue;
					}

					dmem_ptr[dmem_index] = smem_ptr[i] * alpha + dmem_ptr[dmem_index] * beta;
				}
			}
		}
	}
};

template<
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	class A_DMEM_LOADER,
	class B_DMEM_LOADER,
	class C_DMEM_STORER,
	unsigned BLOCK_SIZE,
	class TC_T
	>
__global__ void shgemm_kernel(
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const float alpha,
		const float* const a_ptr, const std::size_t lda,
		const half * const b_ptr, const std::size_t ldb,
		const float beta,
		float* const c_ptr, const std::size_t ldc
		) {
	constexpr unsigned NUM_STAGES = 2;

	extern __shared__ float smem[];
	float* const a_smem_ptr = smem;
	float* const c_smem_ptr = smem + NUM_STAGES * SMEM_M * SMEM_K;
	half * const b_smem_ptr = reinterpret_cast<half*>(c_smem_ptr + SMEM_M * SMEM_N);

	mem_fill_zero<SMEM_M * SMEM_N, BLOCK_SIZE>(c_smem_ptr);

	A_DMEM_LOADER a_dram_loader;
	B_DMEM_LOADER b_dram_loader;

	for (std::size_t block_k = 0; block_k < k; block_k += SMEM_K) {
		a_dram_loader(a_smem_ptr,
				blockIdx.x * SMEM_M, block_k,
				m, k,
				a_ptr, lda
				);
		b_dram_loader(b_smem_ptr,
				block_k, blockIdx.y * SMEM_N,
				k, n,
				b_ptr, ldb
				);
		__syncthreads();

		shgemm_core<SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, TC_T>(c_smem_ptr, a_smem_ptr, b_smem_ptr);
		__syncthreads();
	}

	__syncthreads();
	C_DMEM_STORER c_dmem_storer;
	c_dmem_storer(c_ptr, ldc,
			blockIdx.x * SMEM_M, blockIdx.y * SMEM_N,
			m, n,
			c_smem_ptr,
			alpha, beta);
}

template <class T>
constexpr unsigned size_of = 0;
template <> constexpr unsigned size_of<float> = 4;
template <> constexpr unsigned size_of<half > = 2;

constexpr unsigned get_shared_memory_size_in_byte(
		const unsigned NUM_STAGES,
		const unsigned SMEM_M,
		const unsigned SMEM_N,
		const unsigned SMEM_K
		) {
	return NUM_STAGES * SMEM_M * SMEM_K * size_of<float> +
		NUM_STAGES * SMEM_K * SMEM_N * size_of<half> +
		SMEM_M * SMEM_N * size_of<float>;
}

void shgemm_tn(
		const mtk::shgemm::shgemmHandle_t handle,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const float* const alpha_ptr,
		const float* const a_ptr, const std::size_t lda,
		const half * const b_ptr, const std::size_t ldb,
		const float* const beta_ptr,
		float* const c_ptr, const std::size_t ldc
		) {
	constexpr unsigned NUM_STAGES = 2;
	constexpr unsigned SMEM_M = 64;
	constexpr unsigned SMEM_N = 64;
	constexpr unsigned SMEM_K = 64;
	constexpr unsigned FRAG_M = 16;
	constexpr unsigned FRAG_N = 32;
	constexpr unsigned FRAG_K = 64;
	constexpr unsigned BLOCK_SIZE = 256;
	using TC_T = half;

	constexpr auto smem_size = get_shared_memory_size_in_byte(NUM_STAGES, SMEM_M, SMEM_N, SMEM_K);
	const dim3 grid_size((m + SMEM_M - 1) / SMEM_M, (n + SMEM_N - 1) / SMEM_N);
	const dim3 block_size(BLOCK_SIZE);

	CUTF_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
				&(shgemm_kernel<
					SMEM_M), SMEM_N, SMEM_K,
					FRAG_M, FRAG_N, FRAG_K,
					dmem_loader_n<float, SMEM_K, SMEM_M, BLOCK_SIZE>,
					dmem_loader_n<half , SMEM_K, SMEM_N, BLOCK_SIZE>,
					dmem_storer_n<float, SMEM_M, SMEM_N, BLOCK_SIZE>,
					BLOCK_SIZE,
					TC_T
					>)
				, hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));

	shgemm_kernel<
		SMEM_M, SMEM_N, SMEM_K,
		FRAG_M, FRAG_N, FRAG_K,
		dmem_loader_n<float, SMEM_K, SMEM_M, BLOCK_SIZE>,
		dmem_loader_n<half , SMEM_K, SMEM_N, BLOCK_SIZE>,
		dmem_storer_n<float, SMEM_M, SMEM_N, BLOCK_SIZE>,
		BLOCK_SIZE,
		TC_T
	>
		<<<grid_size, block_size, smem_size, handle.hip_stream>>>
		(
		 m, n, k,
		 *alpha_ptr,
		 a_ptr, lda,
		 b_ptr, ldb,
		 *beta_ptr,
		 c_ptr, ldc
		 );
}
} // noname namespace

void mtk::shgemm::create(
		mtk::shgemm::shgemmHandle_t &handle
		) {
	handle.hip_stream = 0;
}

void mtk::shgemm::set_cuda_stream(
		mtk::shgemm::shgemmHandle_t &handle,
		hipStream_t const hip_stream
		) {
	handle.hip_stream = hip_stream;
}

void mtk::shgemm::shgemm(
		const mtk::shgemm::shgemmHandle_t handle,
		const mtk::shgemm::operation_t op_a,
		const mtk::shgemm::operation_t op_b,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const float* const alpha_ptr,
		const float* const a_ptr, const std::size_t lda,
		const half * const b_ptr, const std::size_t ldb,
		const float* const beta_ptr,
		float* const c_ptr, const std::size_t ldc
		) {
	if (op_a == mtk::shgemm::op_t && op_b == mtk::shgemm::op_n) {
		shgemm_tn(
				handle,
				m, n, k,
				alpha_ptr,
				a_ptr, lda,
				b_ptr, ldb,
				beta_ptr,
				c_ptr, ldc
				);
	}
}
