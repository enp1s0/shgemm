#include "hip/hip_runtime.h"
#include <shgemm/shgemm.hpp>
#include <cutf/cuda.hpp>
#include <cutf/cp_async.hpp>
#include <wmma_extension/tcec/tcec.hpp>
#include <cassert>
#include "shgemm_core.hpp"
#include "dmem_accessor.hpp"

namespace {
template <unsigned SIZE, unsigned BLOCK_SIZE>
__device__ void mem_fill_zero(
		float* const ptr
		) {
	for (unsigned i = 0; i < SIZE; i += BLOCK_SIZE) {
		ptr[i + threadIdx.x] = 0.f;
	}
}

template<
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned FRAG_M,
	unsigned FRAG_N,
	unsigned FRAG_K,
	class A_DMEM_LOADER,
	class B_DMEM_LOADER,
	class C_DMEM_STORER,
	class SHGEMM_CORE,
	unsigned BLOCK_SIZE,
	class TC_T
	>
__global__ void shgemm_kernel(
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const float alpha,
		const float* const a_ptr, const std::size_t lda,
		const half * const b_ptr, const std::size_t ldb,
		const float beta,
		float* const c_ptr, const std::size_t ldc
		) {
	constexpr unsigned NUM_STAGES = 2;

	extern __shared__ float smem[];
	float* const a_smem_ptr = smem;
	float* const c_smem_ptr = smem + NUM_STAGES * SMEM_M * SMEM_K;
	half * const b_smem_ptr = reinterpret_cast<half*>(c_smem_ptr + SMEM_M * SMEM_N);

	mem_fill_zero<SMEM_M * SMEM_N, BLOCK_SIZE>(c_smem_ptr);

	A_DMEM_LOADER a_dram_loader;
	B_DMEM_LOADER b_dram_loader;
	SHGEMM_CORE shgemm_core;

	std::size_t block_k = 0;
	a_dram_loader(a_smem_ptr,
			block_k, blockIdx.y * SMEM_M,
			k, m,
			a_ptr, lda
			);
	b_dram_loader(b_smem_ptr,
			block_k, blockIdx.x * SMEM_N,
			k, n,
			b_ptr, ldb
			);
	block_k += SMEM_K;
	cutf::cp_async::wait_all();
	__syncthreads();

	for (; block_k < k; block_k += SMEM_K) {
		a_dram_loader(a_smem_ptr + ((block_k / SMEM_K) & 0x1) * SMEM_K * SMEM_M,
				block_k, blockIdx.y * SMEM_M,
				k, m,
				a_ptr, lda
				);
		b_dram_loader(b_smem_ptr + ((block_k / SMEM_K) & 0x1) * SMEM_K * SMEM_N,
				block_k, blockIdx.x * SMEM_N,
				k, n,
				b_ptr, ldb
				);

		shgemm_core(c_smem_ptr,
				a_smem_ptr + (1 - ((block_k / SMEM_K) & 0x1)) * SMEM_K * SMEM_M,
				b_smem_ptr + (1 - ((block_k / SMEM_K) & 0x1)) * SMEM_K * SMEM_N
				);
	cutf::cp_async::wait_all();
		__syncthreads();
	}

	shgemm_core(c_smem_ptr,
			a_smem_ptr + (1 - ((block_k / SMEM_K) & 0x1)) * SMEM_K * SMEM_M,
			b_smem_ptr + (1 - ((block_k / SMEM_K) & 0x1)) * SMEM_K * SMEM_N
			);

	__syncthreads();
	C_DMEM_STORER c_dmem_storer;
	c_dmem_storer(c_ptr, ldc,
			blockIdx.y * SMEM_M, blockIdx.x * SMEM_N,
			m, n,
			c_smem_ptr,
			alpha, beta);
}

template <class T>
constexpr unsigned size_of = 0;
template <> constexpr unsigned size_of<float> = 4;
template <> constexpr unsigned size_of<half > = 2;

constexpr unsigned get_shared_memory_size_in_byte(
		const unsigned NUM_STAGES,
		const unsigned SMEM_M,
		const unsigned SMEM_N,
		const unsigned SMEM_K
		) {
	return NUM_STAGES * SMEM_M * SMEM_K * size_of<float> +
		NUM_STAGES * SMEM_K * SMEM_N * size_of<half> +
		SMEM_M * SMEM_N * size_of<float>;
}

void shgemm_tn(
		const mtk::shgemm::shgemmHandle_t handle,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const float* const alpha_ptr,
		const float* const a_ptr, const std::size_t lda,
		const half * const b_ptr, const std::size_t ldb,
		const float* const beta_ptr,
		float* const c_ptr, const std::size_t ldc
		) {
	constexpr unsigned NUM_STAGES = 2;
	constexpr unsigned SMEM_M = 128;
	constexpr unsigned SMEM_N = 64;
	constexpr unsigned SMEM_K = 32;
	constexpr unsigned FRAG_M = 32;
	constexpr unsigned FRAG_N = 64;
	constexpr unsigned FRAG_K = 16;
	constexpr unsigned BLOCK_SIZE = 128;
	using TC_T = half;

	constexpr auto smem_size = get_shared_memory_size_in_byte(NUM_STAGES, SMEM_M, SMEM_N, SMEM_K);
	const dim3 grid_size((n + SMEM_N - 1) / SMEM_N, (m + SMEM_M - 1) / SMEM_M);
	const dim3 block_size(BLOCK_SIZE);

	CUTF_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
				&(shgemm_kernel<
					SMEM_M), SMEM_N, SMEM_K,
					FRAG_M, FRAG_N, FRAG_K,
					mtk::shgemm::device::dmem_loader_n<float, SMEM_K, SMEM_M, BLOCK_SIZE>,
					mtk::shgemm::device::dmem_loader_n<half , SMEM_K, SMEM_N, BLOCK_SIZE>,
					mtk::shgemm::device::dmem_storer_n<float, SMEM_M, SMEM_N, BLOCK_SIZE>,
					mtk::shgemm::device::shgemm_core<SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, TC_T>,
					BLOCK_SIZE,
					TC_T
					>)
				, hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));

	shgemm_kernel<
		SMEM_M, SMEM_N, SMEM_K,
		FRAG_M, FRAG_N, FRAG_K,
		mtk::shgemm::device::dmem_loader_n<float, SMEM_K, SMEM_M, BLOCK_SIZE>,
		mtk::shgemm::device::dmem_loader_n<half , SMEM_K, SMEM_N, BLOCK_SIZE>,
		mtk::shgemm::device::dmem_storer_n<float, SMEM_M, SMEM_N, BLOCK_SIZE>,
		mtk::shgemm::device::shgemm_core<SMEM_M, SMEM_N, SMEM_K, FRAG_M, FRAG_N, FRAG_K, BLOCK_SIZE, TC_T>,
		BLOCK_SIZE,
		TC_T
	>
		<<<grid_size, block_size, smem_size, handle.hip_stream>>>
		(
		 m, n, k,
		 *alpha_ptr,
		 a_ptr, lda,
		 b_ptr, ldb,
		 *beta_ptr,
		 c_ptr, ldc
		 );
}
} // noname namespace

void mtk::shgemm::create(
		mtk::shgemm::shgemmHandle_t &handle
		) {
	handle.hip_stream = 0;
}

void mtk::shgemm::destroy(
		mtk::shgemm::shgemmHandle_t &handle
		) {
}

void mtk::shgemm::set_cuda_stream(
		mtk::shgemm::shgemmHandle_t &handle,
		hipStream_t const hip_stream
		) {
	handle.hip_stream = hip_stream;
}

void mtk::shgemm::shgemm(
		const mtk::shgemm::shgemmHandle_t handle,
		const mtk::shgemm::operation_t op_a,
		const mtk::shgemm::operation_t op_b,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const float* const alpha_ptr,
		const float* const a_ptr, const std::size_t lda,
		const half * const b_ptr, const std::size_t ldb,
		const float* const beta_ptr,
		float* const c_ptr, const std::size_t ldc
		) {
	if (op_a == mtk::shgemm::op_t && op_b == mtk::shgemm::op_n) {
		shgemm_tn(
				handle,
				m, n, k,
				alpha_ptr,
				a_ptr, lda,
				b_ptr, ldb,
				beta_ptr,
				c_ptr, ldc
				);
	}
}
