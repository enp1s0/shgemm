#include <shgemm/shgemm.hpp>

void mtk::shgemm::set_cuda_stream(
		mtk::shgemm::shgemmHandle_t &handle,
		hipStream_t const hip_stream
		) {
	handle.hip_stream = hip_stream;
}

void mtk::shgemm::shgemm(
		const mtk::shgemm::shgemmHandle_t handle,
		const mtk::shgemm::operation_t op_a,
		const mtk::shgemm::operation_t op_b,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const float* const alpha_ptr,
		const float* const a_ptr, const std::size_t lda,
		const half * const b_ptr, const std::size_t ldb,
		const float* const beta_ptr,
		const float* const c_ptr, const std::size_t ldv
		) {

}
